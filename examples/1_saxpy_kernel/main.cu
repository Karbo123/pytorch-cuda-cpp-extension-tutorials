#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <torch/extension.h>

#define CHECK_CUDA(x) TORCH_CHECK(x.is_cuda(), #x " must be a CUDA tensor")
#define CHECK_CONTIGUOUS(x) TORCH_CHECK(x.is_contiguous(), #x " must be contiguous")
#define CHECK_FLOAT32(x) TORCH_CHECK(x.dtype()==torch::kFloat32, #x " must be float32")
#define CHECK_1DARRAY(x) TORCH_CHECK(x.ndimension()==1, #x " must be 1d array")
#define CHECK_INPUT(x) CHECK_CUDA(x); CHECK_CONTIGUOUS(x); CHECK_FLOAT32(x); CHECK_1DARRAY(x)

__global__ void cuSaxpy(float* x, float* y, float* z, 
                        float a, const int size)
{
    const int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < size)
    {
        z[i] = a * x[i] + y[i];
    }
}

torch::Tensor saxpy(torch::Tensor x, torch::Tensor y, float a)
{
    CHECK_INPUT(x); CHECK_INPUT(y);
    TORCH_CHECK(x.size(0) == y.size(0), "size mismatch");

    auto z = torch::empty_like(x);
    const int size = x.size(0);
    const int threads = std::min(size, 1024);
    const int blocks  = std::ceil(size / 1024.0f);
    cuSaxpy <<<blocks, threads>>> (x.data_ptr<float>(), y.data_ptr<float>(), z.data_ptr<float>(), a, size);
    return z;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("saxpy", &saxpy, "SAXPY (CUDA)");
}
